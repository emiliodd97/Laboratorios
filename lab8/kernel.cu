#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdio.h>

#define TX 32
#define TY 32
#define RAD 1

/*  
int divUp(int a, int b){
	return (a + b - 1)/b;
}
*/
// clip values to [0 , 255]
__device__ unsigned char clip(int n){
  return n > 255 ? 255 : (n < 0 ? 0 : n);
}

// bound index values to max size
__device__ int idxClip(int idx, int idxMax){
 return idx >(idxMax - 1) ? (idxMax - 1) : (idx < 0 ? 0 : idx);
}

// transform img(c,r) to flat index i
__device__ int flatten(int col, int row, int width, int height) {
  return idxClip(col, width) + idxClip(row, height)*width;
}


__global__ void filter_kernel(unsigned char *d_input, unsigned char *d_output, int rows, int cols, float* d_window, int window_size) {
  //definición memoria
  extern __shared__ float smem[];
  //shared id (radius)
  const int s_idx = threadIdx.x + RAD;

  //define image row, col position
  const int c = threadIdx.x + blockDim.x * blockIdx.x;
  const int r = threadIdx.y + blockDim.y * blockIdx.y;

  // exit if out of image bounds
  if((c >= cols) || (r >= rows)) return;

  // compute flat index
  const int i = flatten(c, r, cols, rows);
  //cuadrado que respresenta al pixel
  smem[s_idx] = d_input[i];
 
  float pixel_result = 0;

  // apply the convolution window to the image
  // -----------------------------------------
  for(int rd = -RAD; rd <= RAD; ++rd){
    for(int cd = -RAD; cd <= RAD; ++cd){
      
      // compute image and window indexes
      int imgIdx = flatten(c + cd, r + rd, cols, rows);
      int fltIdx = flatten(RAD + cd, RAD + rd, window_size, window_size);

      // read pixel value and window constant
      uchar pixel_val = d_input[imgIdx];
      float weight = d_window[fltIdx];

      // accumulate value
      pixel_result += pixel_val * weight;
    }
  }
 
 // bound pixel value to [0 255] and store in output
  d_output[i] = clip((uchar) pixel_result);
}


// wrapper function definition
void filter_gpu(Mat input, Mat output){

  // define convoltion window size
  const int window_size = 2 * RAD + 1;

  // define convoltion window values

 /*
    const float sharpen[9] = {-1.0, -1.0, -1.0,
                             -1.0,  9.0, -1.0,
                             -1.0, -1.0, -1.0};
  
   const float gaussianBlur[9] = {1/16, 2/16, 1/16,
                                  2/16, 4/16, 2/16,
                                  1/16, 2/16, 1/16};
*/
  const float edgeDetect[9] = {-1, -1, -1,
                               -1,  8, -1,
                               -1, -1, -1};

  unsigned char *inputPtr = (unsigned char*) input.data;
  unsigned char *outputPtr = (unsigned char*) output.data;
  unsigned int cols = input.cols;
  unsigned int rows = input.rows;

  // grid size dimensions (blocks)
  int Bx = (TX + cols -1)/TX;
  int By = (TY + rows -1)/TY;

  // declare pointers to device memory
  unsigned char *d_in  = 0;
  unsigned char *d_out = 0;
  float *d_window = 0;

  // allocate input and output memory in device
  hipMalloc(&d_in, cols*rows*sizeof(unsigned char));
  hipMalloc(&d_out, cols*rows*sizeof(unsigned char));
  hipMalloc(&d_window, window_size*window_size * sizeof(float));

  // copy input data and filter element from host to device
  hipMemcpy(d_in, inputPtr, cols*rows*sizeof(unsigned char), hipMemcpyHostToDevice);
  hipMemcpy(d_window, edgeDetect, window_size*window_size*sizeof(float), hipMemcpyHostToDevice);

  // prepare kernel lauch dimensions
  const dim3 blockSize = dim3(TX, TY);
  const dim3 gridSize = dim3(Bx, By);
  //memory size implementation
  const size_t smemSize = ((TX * TY) + (2 * RAD) * sizeof(float));

  // launch kernel in GPU
  filter_kernel<<<gridSize, blockSize, smemSize>>>(d_in, d_out, rows, cols, d_window, window_size);

  // copy output from device to host
  hipMemcpy(outputPtr, d_out, rows*cols*sizeof(unsigned char), hipMemcpyDeviceToHost);

  // free the memory allocated for device arrays
  hipFree(d_in);
  hipFree(d_out);
  hipFree(d_window);

}
